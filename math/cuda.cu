#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipblas.h>

#include "cpu.h"  // SgemmCpu

hipblasHandle_t handle;

int CopyToDevice(std::shared_ptr<Mat>& mat)
{
  size_t len = mat->size_[0] * mat->size_[1];

  hipError_t error =
      hipMalloc((void**)&mat->data_device_, len * sizeof(float));
  if (error != hipSuccess)
  {
    return -1;
  }

  hipblasStatus_t status = hipblasSetVector(len, sizeof(float), &mat->data_[0], 1,
                                          mat->data_device_, 1);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    return -1;
  }

  return 0;
}

int CopyToHost(std::shared_ptr<Mat>& mat)
{
  size_t len = mat->size_[0] * mat->size_[1];

  hipblasStatus_t status = hipblasGetVector(len, sizeof(float), mat->data_device_,
                                          1, &mat->data_[0], 1);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    return -1;
  }

  return 0;
}

__device__ inline float Relu(float x)
{
  return ((x > 0) ? x : 0);
}

__device__ inline float Sigm(float x)
{
  return 1.0f / (1.0f + __expf(-x));
}

__device__ inline float Tanh(float x)
{
  // return (1.0f - __expf(-x)) / (1.0f + __expf(-x));
  return 1.0f - 2.0f / (__expf(2.0f * x) + 1.0f);
}

__device__ inline float ReluDeriv(float y)
{
  return ((y > 0) ? 1 : 0);
}

__device__ inline float SigmDeriv(float y)
{
  return y * (1 - y);
}

__device__ inline float TanhDeriv(float y)
{
  return 1 - y * y;
}

__global__ void kRelu(float* mat, float* out, unsigned int len)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += num_threads)
  {
    out[i] = Relu(mat[i]);
  }
}

__global__ void kSigm(float* mat, float* out, unsigned int len)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += num_threads)
  {
    out[i] = Sigm(mat[i]);
  }
}

__global__ void kTanh(float* mat, float* out, unsigned int len)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += num_threads)
  {
    out[i] = Tanh(mat[i]);
  }
}

__global__ void kReluDeriv(float* mat1, float* mat2, float* out,
                           unsigned int num_elems)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < num_elems; i += num_threads)
  {
    out[i] = mat1[i] * ReluDeriv(mat2[i]);
  }
}

__global__ void kSigmDeriv(float* mat1, float* mat2, float* out,
                           unsigned int num_elems)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < num_elems; i += num_threads)
  {
    out[i] = mat1[i] * SigmDeriv(mat2[i]);
  }
}

__global__ void kTanhDeriv(float* mat1, float* mat2, float* out,
                           unsigned int num_elems)
{
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int num_threads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < num_elems; i += num_threads)
  {
    out[i] = mat1[i] * TanhDeriv(mat2[i]);
  }
}

#define NUM_BLOCKS 4096
#define NUM_THREADS 512

void MathCuda::Init()
{
  hipblasStatus_t status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    return;
  }

  hipSetDevice(gpu_id_);
}

void MathCuda::Deinit()
{
  hipblasDestroy(handle);
  hipDeviceReset();
}

int MathCuda::Mul(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                  std::shared_ptr<Mat>& out)
{
  int m = mat1->size_[0];
  int k2 = mat1->size_[1];
  int k = mat2->size_[0];
  int n = mat2->size_[1];
  int m2 = out->size_[0];
  int n2 = out->size_[1];
  if (m != m2 || n != n2 || k != k2)
  {
    printf("%d %d %d %d %d %d\n", m, k2, k, n, m2, n2);
    return -1;
  }

  float alpha = 1.0f;
  float beta = 0.0f;

  // Process small matrices on cpu.
  if (m == 1 || n == 1 || k == 1)
  {
    SgemmCpu(true, false, false, m, n, k, alpha, &mat1->data_[0],
             mat1->size_[1], &mat2->data_[0], mat2->size_[1], beta,
             &out->data_[0], out->size_[1]);
  }
  else
  {
    CopyToDevice(mat1);
    CopyToDevice(mat2);
    CopyToDevice(out);

    hipblasStatus_t status =
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                    mat2->data_device_, mat2->size_[1], mat1->data_device_,
                    mat1->size_[1], &beta, out->data_device_, out->size_[1]);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      return -1;
    }

    CopyToHost(out);

    hipFree(out->data_device_);
    hipFree(mat2->data_device_);
    hipFree(mat1->data_device_);
  }

  return 0;
}

int MathCuda::Add(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                  std::shared_ptr<Mat>& out)
{
  int m = mat1->size_[0];
  int k = mat2->size_[0];
  int n = mat2->size_[1];

  float alpha = 1.0f;
  float beta = 0.0f;

  // Process small matrices on cpu.
  if (m == 1 || n == 1 || k == 1)
  {
    for (int i = 0; i < mat1->data_.size(); i++)
    {
      out->data_[i] = mat1->data_[i] + mat2->data_[i];
    }
  }
  else
  {
    CopyToDevice(mat1);
    CopyToDevice(mat2);
    CopyToDevice(out);

    hipblasStatus_t status = hipblasSgeam(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha, mat1->data_device_,
        mat1->size_[1], &beta, mat2->data_device_, mat2->size_[1],
        out->data_device_, out->size_[1]);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      return -1;
    }

    CopyToHost(out);

    hipFree(out->data_device_);
    hipFree(mat2->data_device_);
    hipFree(mat1->data_device_);
  }

  return 0;
}

int MathCuda::ElmtMul(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                      std::shared_ptr<Mat>& out)
{
  CopyToDevice(mat1);
  CopyToDevice(mat2);
  CopyToDevice(out);

  int len = mat1->size_[0] * mat1->size_[1];

  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasStatus_t status = hipblasSgbmv(
      handle, HIPBLAS_OP_N, len, len, 0, 0, &alpha, mat1->data_device_, 1,
      mat2->data_device_, 1, &beta, out->data_device_, 1);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    return -1;
  }

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat2->data_device_);
  hipFree(mat1->data_device_);

  return 0;
}

int MathCuda::AddDeriv(std::shared_ptr<Mat>& mat1d, std::shared_ptr<Mat>& mat2d,
                       std::shared_ptr<Mat>& out)
{
  for (int i = 0; i < mat1d->data_.size(); i++)
  {
    float curr = out->data_[i];
    mat1d->data_[i] += curr;
    mat2d->data_[i] += curr;
  }
  return 0;
}

int MathCuda::ElmtMulDeriv(std::shared_ptr<Mat>& mat1,
                           std::shared_ptr<Mat>& mat2,
                           std::shared_ptr<Mat>& mat1d,
                           std::shared_ptr<Mat>& mat2d,
                           std::shared_ptr<Mat>& out)
{
  for (int i = 0; i < mat1->data_.size(); i++)
  {
    float curr = out->data_[i];
    mat1d->data_[i] += mat2->data_[i] * curr;
    mat2d->data_[i] += mat1->data_[i] * curr;
  }
  return 0;
}

int MathCuda::MulDeriv(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                       std::shared_ptr<Mat>& mat1d, std::shared_ptr<Mat>& mat2d,
                       std::shared_ptr<Mat>& out)
{
  int mat1_size1 = mat1->size_[1];
  int mat2_size1 = mat2->size_[1];
  for (int i = 0; i < mat1->size_[0]; i++)
  {  // loop over rows of m1
    for (int j = 0; j < mat2_size1; j++)
    {  // loop over cols of m2
      for (int k = 0; k < mat1_size1; k++)
      {  // dot product loop
        float b = out->data_[mat2_size1 * i + j];
        mat1d->data_[mat1_size1 * i + k] += mat2->data_[mat2_size1 * k + j] * b;
        mat2d->data_[mat2_size1 * k + j] += mat1->data_[mat1_size1 * i + k] * b;
      }
    }
  }
  return 0;
}

int MathCuda::Relu(std::shared_ptr<Mat>& mat, std::shared_ptr<Mat>& out)
{
  unsigned int len = mat->size_[0] * mat->size_[1];

  CopyToDevice(mat);
  CopyToDevice(out);

  if (mat->size_[0] != out->size_[0] || mat->size_[1] != out->size_[1])
  {
    return -1;
  }

  kRelu << <NUM_BLOCKS, NUM_THREADS>>>
      (mat->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat->data_device_);

  return 0;
}

int MathCuda::Sigm(std::shared_ptr<Mat>& mat, std::shared_ptr<Mat>& out)
{
  unsigned int len = mat->size_[0] * mat->size_[1];

  CopyToDevice(mat);
  CopyToDevice(out);

  if (mat->size_[0] != out->size_[0] || mat->size_[1] != out->size_[1])
  {
    return -1;
  }

  kSigm << <NUM_BLOCKS, NUM_THREADS>>>
      (mat->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat->data_device_);

  return 0;
}

int MathCuda::Tanh(std::shared_ptr<Mat>& mat, std::shared_ptr<Mat>& out)
{
  unsigned int len = mat->size_[0] * mat->size_[1];

  CopyToDevice(mat);
  CopyToDevice(out);

  if (mat->size_[0] != out->size_[0] || mat->size_[1] != out->size_[1])
  {
    return -1;
  }

  kTanh << <NUM_BLOCKS, NUM_THREADS>>>
      (mat->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat->data_device_);

  return 0;
}

int MathCuda::ReluDeriv(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                        std::shared_ptr<Mat>& out)
{
  int len = mat1->size_[0] * mat1->size_[1];

  CopyToDevice(mat1);
  CopyToDevice(mat2);
  CopyToDevice(out);

  if (mat1->size_[0] != mat2->size_[0] || mat1->size_[1] != mat2->size_[1] ||
      mat1->size_[0] != out->size_[0] || mat1->size_[1] != out->size_[1])
  {
    return -1;
  }

  kReluDeriv << <NUM_BLOCKS, NUM_THREADS>>>
      (mat1->data_device_, mat2->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat2->data_device_);
  hipFree(mat1->data_device_);

  return 0;
}

int MathCuda::SigmDeriv(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                        std::shared_ptr<Mat>& out)
{
  int len = mat1->size_[0] * mat1->size_[1];

  CopyToDevice(mat1);
  CopyToDevice(mat2);
  CopyToDevice(out);

  if (mat1->size_[0] != mat2->size_[0] || mat1->size_[1] != mat2->size_[1] ||
      mat1->size_[0] != out->size_[0] || mat1->size_[1] != out->size_[1])
  {
    return -1;
  }

  kSigmDeriv << <NUM_BLOCKS, NUM_THREADS>>>
      (mat1->data_device_, mat2->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat2->data_device_);
  hipFree(mat1->data_device_);

  return 0;
}

int MathCuda::TanhDeriv(std::shared_ptr<Mat>& mat1, std::shared_ptr<Mat>& mat2,
                        std::shared_ptr<Mat>& out)
{
  int len = mat1->size_[0] * mat1->size_[1];

  CopyToDevice(mat1);
  CopyToDevice(mat2);
  CopyToDevice(out);

  if (mat1->size_[0] != mat2->size_[0] || mat1->size_[1] != mat2->size_[1] ||
      mat1->size_[0] != out->size_[0] || mat1->size_[1] != out->size_[1])
  {
    return -1;
  }

  kTanhDeriv << <NUM_BLOCKS, NUM_THREADS>>>
      (mat1->data_device_, mat2->data_device_, out->data_device_, len);

  CopyToHost(out);

  hipFree(out->data_device_);
  hipFree(mat2->data_device_);
  hipFree(mat1->data_device_);

  return 0;
}
