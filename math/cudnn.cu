#include "hipDNN.h"

#include <stdio.h>
#include <hipblas.h>
#include <hipDNN.h>
#include <float.h>

#include "cpu.h"  // SgemmCpu, default implementation

using std::string;
using std::vector;
using std::shared_ptr;

static shared_ptr<MathCpu> math_cpu(new MathCpu);

const hipdnnDataType_t data_type = HIPDNN_DATA_FLOAT;
const hipdnnTensorFormat_t tensor_format = HIPDNN_TENSOR_NCHW;

hipdnnTensorDescriptor_t descr_tensor_src, descr_tensor_dst, descr_tensor_bias;
hipdnnFilterDescriptor_t descr_filter;
hipdnnConvolutionDescriptor_t descr_conv;
hipdnnPoolingDescriptor_t descr_pooling;
hipdnnHandle_t cudnn_handle;
hipblasHandle_t cublas_handle;

inline void CheckCuda(hipError_t status)
{
  if (status != 0)
  {
    printf("cuda error: %s\n", hipGetErrorString(status));
  }
}

inline void CheckCudnn(hipdnnStatus_t status)
{
  if (status != HIPDNN_STATUS_SUCCESS)
  {
    printf("cudnn error: %s\n", hipdnnGetErrorString(status));
  }
}

inline void CheckCublas(int status)
{
  if (status != 0)
  {
    printf("cublas error: %u\n", status);
  }
}

inline static int CopyToDevice(shared_ptr<Mat> &mat)
{
  size_t len = mat->size_[0] * mat->size_[1] * mat->size_[2] * mat->size_[3];

  if (mat->data_device_ == nullptr)
  {
    hipError_t error =
        hipMalloc((void **)&mat->data_device_, len * sizeof(float));
    if (error != hipSuccess)
    {
      return -1;
    }
  }

  CheckCublas(hipblasSetVector(len, sizeof(float), &mat->data_[0], 1,
                              mat->data_device_, 1));

  return 0;
}

inline static int CopyToHost(shared_ptr<Mat> &mat)
{
  size_t len = mat->size_[0] * mat->size_[1] * mat->size_[2] * mat->size_[3];

  CheckCublas(hipblasGetVector(len, sizeof(float), mat->data_device_, 1,
                              &mat->data_[0], 1));

  return 0;
}

void MathCudnn::Init()
{
  CheckCudnn(hipdnnCreate(&cudnn_handle));
  CheckCudnn(hipdnnCreateTensorDescriptor(&descr_tensor_src));
  CheckCudnn(hipdnnCreateTensorDescriptor(&descr_tensor_dst));
  CheckCudnn(hipdnnCreateTensorDescriptor(&descr_tensor_bias));
  CheckCudnn(hipdnnCreateFilterDescriptor(&descr_filter));
  CheckCudnn(hipdnnCreateConvolutionDescriptor(&descr_conv));
  CheckCudnn(hipdnnCreatePoolingDescriptor(&descr_pooling));

  CheckCublas(hipblasCreate(&cublas_handle));

  hipSetDevice(0);
}

void MathCudnn::Deinit()
{
  CheckCublas(hipblasDestroy(cublas_handle));

  CheckCudnn(hipdnnDestroyPoolingDescriptor(descr_pooling));
  CheckCudnn(hipdnnDestroyConvolutionDescriptor(descr_conv));
  CheckCudnn(hipdnnDestroyFilterDescriptor(descr_filter));
  CheckCudnn(hipdnnDestroyTensorDescriptor(descr_tensor_src));
  CheckCudnn(hipdnnDestroyTensorDescriptor(descr_tensor_dst));
  CheckCudnn(hipdnnDestroyTensorDescriptor(descr_tensor_bias));
  CheckCudnn(hipdnnDestroy(cudnn_handle));
}

int MathCudnn::FreeMatMemory(float *ptr)
{
  hipFree(ptr);
  return 0;
}

int MathCudnn::Mul(shared_ptr<Mat> &mat1, shared_ptr<Mat> &mat2,
                   shared_ptr<Mat> &out)
{
  int m = mat1->size_[0];
  int k2 = mat1->size_[1];
  int k = mat2->size_[0];
  int n = mat2->size_[1];
  int m2 = out->size_[0];
  int n2 = out->size_[1];
  if (m != m2 || n != n2 || k != k2)
  {
    printf("%d %d %d %d %d %d\n", m, k2, k, n, m2, n2);
    return -1;
  }

  // Process small matrices on cpu.
  if (m == 1 || n == 1 || k == 1)
  {
    math_cpu->Mul(mat1, mat2, out);
  }
  else
  {
    CopyToDevice(mat1);
    CopyToDevice(mat2);
    CopyToDevice(out);

    float alpha = 1.0f;
    float beta = 0.0f;
    CheckCublas(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
                            &alpha, mat2->data_device_, mat2->size_[1],
                            mat1->data_device_, mat1->size_[1], &beta,
                            out->data_device_, out->size_[1]));

    CopyToHost(out);
  }

  return 0;
}

int MathCudnn::Add(shared_ptr<Mat> &mat1, shared_ptr<Mat> &mat2,
                   shared_ptr<Mat> &out)
{
  int m = mat1->size_[0];
  int k = mat2->size_[0];
  int n = mat2->size_[1];

  // Process small matrices on cpu.
  if (m == 1 || n == 1 || k == 1)
  {
    math_cpu->Add(mat1, mat2, out);
  }
  else
  {
    CopyToDevice(mat1);
    CopyToDevice(mat2);
    CopyToDevice(out);

    float alpha = 1.0f;
    float beta = 1.0f;
    hipblasStatus_t status = hipblasSgeam(
        cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha,
        mat1->data_device_, mat1->size_[1], &beta, mat2->data_device_,
        mat2->size_[1], out->data_device_, out->size_[1]);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      return -1;
    }

    CopyToHost(out);
  }

  return 0;
}

int MathCudnn::ElmtMul(shared_ptr<Mat> &mat1, shared_ptr<Mat> &mat2,
                       shared_ptr<Mat> &out)
{
  CopyToDevice(mat1);
  CopyToDevice(mat2);
  CopyToDevice(out);

  int len = mat1->size_[0] * mat1->size_[1] * mat1->size_[2] * mat1->size_[3];

  float alpha = 1.0f;
  float beta = 0.0f;
  hipblasStatus_t status = hipblasSgbmv(
      cublas_handle, HIPBLAS_OP_N, len, len, 0, 0, &alpha, mat1->data_device_, 1,
      mat2->data_device_, 1, &beta, out->data_device_, 1);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    return -1;
  }

  CopyToHost(out);

  return 0;
}

int MathCudnn::AddDeriv(shared_ptr<Mat> &mat1d, shared_ptr<Mat> &mat2d,
                        shared_ptr<Mat> &out)
{
  return math_cpu->AddDeriv(mat1d, mat2d, out);
}

int MathCudnn::ElmtMulDeriv(shared_ptr<Mat> &mat1, shared_ptr<Mat> &mat2,
                            shared_ptr<Mat> &mat1d, shared_ptr<Mat> &mat2d,
                            shared_ptr<Mat> &out)
{
  return math_cpu->ElmtMulDeriv(mat1, mat2, mat1d, mat2d, out);
}

int MathCudnn::MulDeriv(shared_ptr<Mat> &mat1, shared_ptr<Mat> &mat2,
                        shared_ptr<Mat> &mat1d, shared_ptr<Mat> &mat2d,
                        shared_ptr<Mat> &out)
{
  int m = mat1d->size_[0];
  int n = mat1d->size_[1];
  int k = mat2->size_[1];
  float alpha = 1.0f;
  float beta = 0.0f;
  // SgemmCpu(true, false, true, m, n, k, alpha, &out->data_[0], k,
  //         &mat2->data_[0], k, beta, &mat1d->data_[0], n);

  CopyToDevice(mat1);
  CopyToDevice(mat1d);
  CopyToDevice(mat2);
  CopyToDevice(mat2d);
  CopyToDevice(out);

  CheckCublas(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                          &alpha, mat2->data_device_, k, out->data_device_,
                          out->size_[1], &beta, mat1d->data_device_, n));

  m = mat2d->size_[1];
  n = mat1->size_[1];
  k = mat1->size_[0];
  alpha = 1.0f;
  beta = 0.0f;
  // SgemmCpu(false, false, true, m, n, k, alpha, &out->data_[0], m,
  //         &mat1->data_[0], n, beta, &mat2d->data_[0], m);

  CheckCublas(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k,
                          &alpha, out->data_device_, m, mat1->data_device_, n,
                          &beta, mat2d->data_device_, m));

  CopyToHost(mat1d);
  CopyToHost(mat2d);

  return 0;
}

int MathCudnn::Relu(shared_ptr<Mat> &in_w, shared_ptr<Mat> &out_w)
{
  CopyToDevice(in_w);
  CopyToDevice(out_w);

  if (in_w->size_[0] != out_w->size_[0] || in_w->size_[1] != out_w->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationForward(cudnn_handle, HIPDNN_ACTIVATION_RELU, &alpha,
                                    descr_tensor_src, in_w->data_device_, &beta,
                                    descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::Sigm(shared_ptr<Mat> &in_w, shared_ptr<Mat> &out_w)
{
  CopyToDevice(in_w);
  CopyToDevice(out_w);

  if (in_w->size_[0] != out_w->size_[0] || in_w->size_[1] != out_w->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationForward(
      cudnn_handle, HIPDNN_ACTIVATION_SIGMOID, &alpha, descr_tensor_src,
      in_w->data_device_, &beta, descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::Tanh(shared_ptr<Mat> &in_w, shared_ptr<Mat> &out_w)
{
  CopyToDevice(in_w);
  CopyToDevice(out_w);

  if (in_w->size_[0] != out_w->size_[0] || in_w->size_[1] != out_w->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationForward(cudnn_handle, HIPDNN_ACTIVATION_TANH, &alpha,
                                    descr_tensor_src, in_w->data_device_, &beta,
                                    descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::ReluDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                         shared_ptr<Mat> &out_w, shared_ptr<Mat> &out_dw)
{
  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(out_w);
  CopyToDevice(out_dw);

  if (out_dw->size_[0] != out_w->size_[0] ||
      out_dw->size_[1] != out_w->size_[1] ||
      out_dw->size_[0] != in_dw->size_[0] ||
      out_dw->size_[1] != in_dw->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationBackward(
      cudnn_handle, HIPDNN_ACTIVATION_RELU, &alpha, descr_tensor_src,
      out_w->data_device_, descr_tensor_src, out_dw->data_device_,
      descr_tensor_dst, in_w->data_device_, &beta, descr_tensor_dst,
      in_dw->data_device_));

  CopyToHost(in_dw);

  return 0;
}

int MathCudnn::SigmDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                         shared_ptr<Mat> &out_w, shared_ptr<Mat> &out_dw)
{
  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(out_w);
  CopyToDevice(out_dw);

  if (out_dw->size_[0] != out_w->size_[0] ||
      out_dw->size_[1] != out_w->size_[1] ||
      out_dw->size_[0] != in_dw->size_[0] ||
      out_dw->size_[1] != in_dw->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationBackward(
      cudnn_handle, HIPDNN_ACTIVATION_SIGMOID, &alpha, descr_tensor_src,
      out_w->data_device_, descr_tensor_src, out_dw->data_device_,
      descr_tensor_dst, in_w->data_device_, &beta, descr_tensor_dst,
      in_dw->data_device_));

  CopyToHost(in_dw);

  return 0;
}

int MathCudnn::TanhDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                         shared_ptr<Mat> &out_w, shared_ptr<Mat> &out_dw)
{
  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(out_w);
  CopyToDevice(out_dw);

  if (out_dw->size_[0] != out_w->size_[0] ||
      out_dw->size_[1] != out_w->size_[1] ||
      out_dw->size_[0] != in_dw->size_[0] ||
      out_dw->size_[1] != in_dw->size_[1])
  {
    return -1;
  }

  int n = in_w->size_[3];
  int c = in_w->size_[2];
  int h = in_w->size_[1];
  int w = in_w->size_[0];

  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format,
                                        data_type, n, c, h, w));
  CheckCudnn(hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format,
                                        data_type, n, c, h, w));

  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnActivationBackward(
      cudnn_handle, HIPDNN_ACTIVATION_TANH, &alpha, descr_tensor_src,
      out_w->data_device_, descr_tensor_src, out_dw->data_device_,
      descr_tensor_dst, in_w->data_device_, &beta, descr_tensor_dst,
      in_dw->data_device_));

  CopyToHost(in_dw);

  return 0;
}

shared_ptr<Mat> MathCudnn::Softmax(shared_ptr<Mat> &mat)
{
  return math_cpu->Softmax(mat);
}

int MathCudnn::Fc(shared_ptr<Mat> &in, shared_ptr<Mat> &filters,
                  shared_ptr<Mat> &biases, shared_ptr<Mat> &out)
{
  int num_out = out->size_[2];
  int num_in = filters->size_[0];
  int num_batch = in->size_[3];

  std::vector<int> in_size(in->size_);
  std::vector<int> out_size(out->size_);
  in->size_[0] = num_batch;
  in->size_[1] = num_in;
  in->size_[2] = 1;
  in->size_[3] = 1;
  out->size_[0] = num_batch;
  out->size_[1] = num_out;
  out->size_[2] = 1;
  out->size_[3] = 1;
  Mul(in, filters, out);
  in->size_ = in_size;
  out->size_ = out_size;

  // math->Add(biases, out, out);
  for (int batch = 0; batch < num_batch; ++batch)
  {
    int out_offset = num_out * batch;
    for (int i = 0; i < num_out; ++i)
    {
      out->data_[out_offset + i] += biases->data_[i];
    }
  }

  return 0;
}

int MathCudnn::FcDeriv(shared_ptr<Mat> &in, shared_ptr<Mat> &filters,
                       shared_ptr<Mat> &biases, shared_ptr<Mat> &out)
{
  int num_out = out->size_[2];
  int num_in = filters->size_[0];
  int num_batch = in->size_[3];

  std::vector<int> in_size(in->size_);
  std::vector<int> in_dw_size(in->dw_->size_);
  std::vector<int> out_dw_size(out->dw_->size_);
  in->size_[0] = num_batch;
  in->size_[1] = num_in;
  in->size_[2] = 1;
  in->size_[3] = 1;
  in->dw_->size_[0] = num_batch;
  in->dw_->size_[1] = num_in;
  in->dw_->size_[2] = 1;
  in->dw_->size_[3] = 1;
  out->dw_->size_[0] = num_batch;
  out->dw_->size_[1] = num_out;
  out->dw_->size_[2] = 1;
  out->dw_->size_[3] = 1;
  MulDeriv(in, filters, in->dw_, filters->dw_, out->dw_);
  in->size_ = in_size;
  in->dw_->size_ = in_dw_size;
  out->dw_->size_ = out_dw_size;

  // math->AddDeriv(biases->dw_, out->dw_, out->dw_);
  for (int batch = 0; batch < num_batch; ++batch)
  {
    int out_offset = num_out * batch;
    for (int i = 0; i < num_out; ++i)
    {
      float dw = out->dw_->data_[out_offset + i];
      biases->dw_->data_[i] += dw;
    }
  }

  return 0;
}

int MathCudnn::Conv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &filters_w,
                    shared_ptr<Mat> &biases_w, shared_ptr<Mat> &out_w,
                    ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int num_input = in_w->size_[2];
  int num_filters = filters_w->size_[3];
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(filters_w);
  CopyToDevice(biases_w);
  CopyToDevice(out_w);

  int n = batch_size;
  int c = num_input;
  int h = in_height;
  int w = in_width;
  // printf("conv %u %u %u %u\n", n, c, h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);

  static const int kDims = 4;
  const int filterDimA[kDims] = {num_filters, c, filter_height, filter_width};
  CheckCudnn(
      hipdnnSetFilterNdDescriptor(descr_filter, data_type, kDims, filterDimA));

  static const int kConvDims = 2;
  int padding[kConvDims] = {padding_x, padding_y};
  int stride[kConvDims] = {stride_x, stride_y};
  int upscale[kConvDims] = {1, 1};  // _v3 TODO
  CheckCudnn(hipdnnSetConvolutionNdDescriptor(descr_conv, kConvDims, padding,
                                             stride, upscale,
                                             HIPDNN_CROSS_CORRELATION));
  // find dimension of convolution output
  /*int tensor_ouput_dim[tensorDims];
  CheckCudnn(cudnnGetConvolutionNdForwardOutputDim(
      convDesc, srcTensorDesc, filterDesc, tensorDims, tensor_ouput_dim));
  n = tensor_ouput_dim[0];
  c = tensor_ouput_dim[1];
  h = tensor_ouput_dim[2];
  w = tensor_ouput_dim[3];*/

  n = batch_size;
  c = num_filters;
  h = out_height;
  w = out_width;
  // printf("%u %u %u %u\n", n, c, h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_bias, tensor_format, data_type, 1, c,
                             1, 1);

  /*
    // Choose the best algo according to the preference
    cout << "Testing hipdnnGetConvolutionForwardAlgorithm ...\n";
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(
        cudnnHandle, srcTensorDesc, filterDesc, convDesc, dstTensorDesc,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
    cout << "Fastest algorithm is Algo " << algo << "\n";
    convAlgorithm = algo;
    // New way of finding the fastest config
    // Setup for findFastest call
    cout << "Testing hipdnnFindConvolutionForwardAlgorithm ...\n";
    int requestedAlgoCount = 5;
    int returnedAlgoCount[1];
    hipdnnConvolutionFwdAlgoPerf_t* results =
        (hipdnnConvolutionFwdAlgoPerf_t*)malloc(
            sizeof(hipdnnConvolutionFwdAlgoPerf_t) * requestedAlgoCount);
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
        cudnnHandle, srcTensorDesc, filterDesc, convDesc, dstTensorDesc,
        requestedAlgoCount, returnedAlgoCount, results));
    for (int algoIndex = 0; algoIndex < *returnedAlgoCount; ++algoIndex)
    {
      printf("^^^^ %s for Algo %d: %f time requiring %llu memory\n",
             hipdnnGetErrorString(results[algoIndex].status),
             results[algoIndex].algo, results[algoIndex].time,
             (unsigned long long)results[algoIndex].memory);
    }
    free(results);*/

  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  size_t sizeInBytes = 0;
  void *workSpace = NULL;
  CheckCudnn(hipdnnGetConvolutionForwardWorkspaceSize(
      cudnn_handle, descr_tensor_src, descr_filter, descr_conv,
      descr_tensor_dst, algo, &sizeInBytes));
  if (sizeInBytes != 0)
  {
    CheckCuda(hipMalloc(&workSpace, sizeInBytes));
  }

  float alpha = 1;
  float beta = 0;  // 1
  CheckCudnn(hipdnnConvolutionForward(
      cudnn_handle, &alpha, descr_tensor_src, in_w->data_device_, descr_filter,
      filters_w->data_device_, descr_conv, algo, workSpace, sizeInBytes, &beta,
      descr_tensor_dst, out_w->data_device_));
  if (sizeInBytes != 0)
  {
    CheckCuda(hipFree(workSpace));
  }

  float bias_alpha = 1;
  float bias_beta = 1;
  CheckCudnn(hipdnnAddTensor(cudnn_handle, CUDNN_ADD_SAME_C, &bias_alpha,
                            descr_tensor_bias, biases_w->data_device_,
                            &bias_beta, descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::ConvDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                         shared_ptr<Mat> &filters_w,
                         shared_ptr<Mat> &filters_dw,
                         shared_ptr<Mat> &biases_dw, shared_ptr<Mat> &out_w,
                         shared_ptr<Mat> &out_dw, ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int num_input = in_w->size_[2];
  int num_filters = filters_w->size_[3];
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(filters_w);
  CopyToDevice(filters_dw);
  CopyToDevice(biases_dw);
  CopyToDevice(out_dw);

  int n = batch_size;
  int c = num_filters;
  int h = out_height;
  int w = out_width;
  // printf("convderiv %u %u %u %u\n", n, c, h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_bias, tensor_format, data_type, 1, c,
                             1, 1);

  static const int kDims = 4;
  const int filterDimA[kDims] = {num_filters, num_input, filter_height,
                                 filter_width};
  CheckCudnn(
      hipdnnSetFilterNdDescriptor(descr_filter, data_type, kDims, filterDimA));

  static const int kConvDims = 2;
  int padding[kConvDims] = {padding_x, padding_y};
  int stride[kConvDims] = {stride_x, stride_y};
  int upscale[kConvDims] = {1, 1};  // _v3 TODO
  CheckCudnn(hipdnnSetConvolutionNdDescriptor(descr_conv, kConvDims, padding,
                                             stride, upscale,
                                             HIPDNN_CROSS_CORRELATION));

  n = batch_size;
  c = num_input;
  h = in_height;
  w = in_width;
  // printf("%u %u %u %u\n", n, c, h, w);
  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);

  // get workspace for backwards filter algorithm
  hipdnnConvolutionBwdFilterAlgo_t algo_filter =
      HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
  size_t size_in_bytes_f = 0;
  void *work_space_f = NULL;
  CheckCudnn(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
      cudnn_handle, descr_tensor_dst, descr_tensor_src, descr_conv,
      descr_filter, algo_filter, &size_in_bytes_f));
  // printf("filters size: %u\n", sizeInBytesF);
  if (size_in_bytes_f != 0)
  {
    CheckCuda(hipMalloc(&work_space_f, size_in_bytes_f));
  }

  float bias_alpha = 1;
  float bias_beta = 1;
  CheckCudnn(hipdnnConvolutionBackwardBias(
      cudnn_handle, &bias_alpha, descr_tensor_src, out_dw->data_device_,
      &bias_beta, descr_tensor_bias, biases_dw->data_device_));

  float alpha_f = 1;
  float beta_f = 1;
  CheckCudnn(cudnnConvolutionBackwardFilter_v3(
      cudnn_handle, &alpha_f, descr_tensor_dst, in_w->data_device_,
      descr_tensor_src, out_dw->data_device_, descr_conv, algo_filter,
      work_space_f, size_in_bytes_f, &beta_f, descr_filter,
      filters_dw->data_device_));

  if (size_in_bytes_f != 0)
  {
    CheckCuda(hipFree(work_space_f));
  }

  hipdnnConvolutionBwdDataAlgo_t algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  size_t size_in_bytes = 0;
  void *work_space = NULL;
  CheckCudnn(hipdnnGetConvolutionBackwardDataWorkspaceSize(
      cudnn_handle, descr_filter, descr_tensor_src, descr_conv,
      descr_tensor_dst, algo, &size_in_bytes));
  // printf("data size: %u\n", sizeInBytes);
  if (size_in_bytes != 0)
  {
    CheckCuda(hipMalloc(&work_space, size_in_bytes));
  }

  float alpha = 1;
  float beta = 0;  // 1
  CheckCudnn(cudnnConvolutionBackwardData_v3(
      cudnn_handle, &alpha, descr_filter, filters_w->data_device_,
      descr_tensor_src, out_dw->data_device_, descr_conv, algo, work_space,
      size_in_bytes, &beta, descr_tensor_dst, in_dw->data_device_));
  if (size_in_bytes != 0)
  {
    CheckCuda(hipFree(work_space));
  }

  CopyToHost(in_dw);
  CopyToHost(filters_dw);
  CopyToHost(biases_dw);

  return 0;
}

int MathCudnn::MaxPool(shared_ptr<Mat> &in_w, shared_ptr<Mat> &out_w,
                       ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int num_filters = in_w->size_[2];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(out_w);

  int n = batch_size;
  int c = num_filters;
  int h = in_height;
  int w = in_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  static const int kDims = 2;
  int filter_dim[kDims] = {filter_width, filter_height};
  int padding[kDims] = {padding_x, padding_y};
  int stride[kDims] = {stride_x, stride_y};
  CheckCudnn(hipdnnSetPoolingNdDescriptor(descr_pooling, HIPDNN_POOLING_MAX,
                                         kDims, filter_dim, padding, stride));

  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);

  /*const int tensorDims = 4;
  int tensor_ouput_dim[tensorDims] = {n, c, h, w};
  checkCUDNN(cudnnGetPoolingNdForwardOutputDim(poolingDesc, srcTensorDesc,
                                               tensorDims, tensor_ouput_dim));
  n = tensor_ouput_dim[0];
  c = tensor_ouput_dim[1];
  h = tensor_ouput_dim[2];
  w = tensor_ouput_dim[3];*/

  n = batch_size;
  c = num_filters;
  h = out_height;
  w = out_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);
  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnPoolingForward(cudnn_handle, descr_pooling, &alpha,
                                 descr_tensor_src, in_w->data_device_, &beta,
                                 descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::MaxPoolDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                            shared_ptr<Mat> &out_w, shared_ptr<Mat> &out_dw,
                            ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int num_filters = in_w->size_[2];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(out_w);
  CopyToDevice(out_dw);

  int n = batch_size;
  int c = num_filters;
  int h = out_height;
  int w = out_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  static const int kDims = 2;
  int filter_dim[kDims] = {filter_width, filter_height};
  int padding[kDims] = {padding_x, padding_y};
  int stride[kDims] = {stride_x, stride_y};
  CheckCudnn(hipdnnSetPoolingNdDescriptor(descr_pooling, HIPDNN_POOLING_MAX,
                                         kDims, filter_dim, padding, stride));

  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);

  n = batch_size;
  c = num_filters;
  h = in_height;
  w = in_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);
  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnPoolingBackward(cudnn_handle, descr_pooling, &alpha,
                                  descr_tensor_src, out_w->data_device_,
                                  descr_tensor_src, out_dw->data_device_,
                                  descr_tensor_dst, in_w->data_device_, &beta,
                                  descr_tensor_dst, in_dw->data_device_));

  CopyToHost(in_dw);

  return 0;
}

int MathCudnn::AvePool(shared_ptr<Mat> &in_w, shared_ptr<Mat> &out_w,
                       ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int num_filters = in_w->size_[2];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(out_w);

  int n = batch_size;
  int c = num_filters;
  int h = in_height;
  int w = in_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  static const int kDims = 2;
  int filter_dim[kDims] = {filter_width, filter_height};
  int padding[kDims] = {padding_x, padding_y};
  int stride[kDims] = {stride_x, stride_y};
  CheckCudnn(hipdnnSetPoolingNdDescriptor(
      descr_pooling, HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING, kDims,
      filter_dim, padding, stride));

  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);

  /*const int tensorDims = 4;
  int tensor_ouput_dim[tensorDims] = {n, c, h, w};
  checkCUDNN(cudnnGetPoolingNdForwardOutputDim(poolingDesc, srcTensorDesc,
                                               tensorDims, tensor_ouput_dim));
  n = tensor_ouput_dim[0];
  c = tensor_ouput_dim[1];
  h = tensor_ouput_dim[2];
  w = tensor_ouput_dim[3];*/

  n = batch_size;
  c = num_filters;
  h = out_height;
  w = out_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);
  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnPoolingForward(cudnn_handle, descr_pooling, &alpha,
                                 descr_tensor_src, in_w->data_device_, &beta,
                                 descr_tensor_dst, out_w->data_device_));

  CopyToHost(out_w);

  return 0;
}

int MathCudnn::AvePoolDeriv(shared_ptr<Mat> &in_w, shared_ptr<Mat> &in_dw,
                            shared_ptr<Mat> &out_w, shared_ptr<Mat> &out_dw,
                            ConvParams &conv_params)
{
  int padding_x = conv_params.padding_x;
  int padding_y = conv_params.padding_y;
  int stride_x = conv_params.stride_x;
  int stride_y = conv_params.stride_y;
  int filter_width = conv_params.filter_width;
  int filter_height = conv_params.filter_height;
  int in_width = in_w->size_[0];
  int in_height = in_w->size_[1];
  int num_filters = in_w->size_[2];
  int batch_size = in_w->size_[3];

  int out_width = (in_width + padding_x * 2 - filter_width) / stride_x + 1;
  int out_height = (in_height + padding_y * 2 - filter_height) / stride_y + 1;

  CopyToDevice(in_w);
  CopyToDevice(in_dw);
  CopyToDevice(out_w);
  CopyToDevice(out_dw);

  int n = batch_size;
  int c = num_filters;
  int h = out_height;
  int w = out_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  static const int kDims = 2;
  int filter_dim[kDims] = {filter_width, filter_height};
  int padding[kDims] = {padding_x, padding_y};
  int stride[kDims] = {stride_x, stride_y};
  CheckCudnn(hipdnnSetPoolingNdDescriptor(
      descr_pooling, HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING, kDims,
      filter_dim, padding, stride));

  hipdnnSetTensor4dDescriptor(descr_tensor_src, tensor_format, data_type, n, c,
                             h, w);

  n = batch_size;
  c = num_filters;
  h = in_height;
  w = in_width;
  // printf("%u %u %u %u\n", n, c, h, w);

  hipdnnSetTensor4dDescriptor(descr_tensor_dst, tensor_format, data_type, n, c,
                             h, w);
  float alpha = 1;
  float beta = 0;
  CheckCudnn(hipdnnPoolingBackward(cudnn_handle, descr_pooling, &alpha,
                                  descr_tensor_src, out_w->data_device_,
                                  descr_tensor_src, out_dw->data_device_,
                                  descr_tensor_dst, in_w->data_device_, &beta,
                                  descr_tensor_dst, in_dw->data_device_));

  CopyToHost(in_dw);

  return 0;
}
